#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

#include<stdio.h>
#include<fstream>
#include<sstream>
#include<string>
#include<vector>
#include<algorithm>
#include<iterator>

using namespace std;

#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__
#endif 

// structure for trajectory
struct trajectory {
	// a struct containing all the points consistent in a trajectory
	vector<int> trajectoryPoints;
	float* distance = new float[0];
};

// global function for the GPU to calculate the total distance in the entire trajectory
__global__ void euclidean(int* points, int stride, float* distArray, float* distance, int length) {
	int i = threadIdx.x;
	int index = i * stride;
	int squaredSum = 0;

	// allow multi-dimensional
	for (int j = 0; j < stride; j++) {
		squaredSum += (points[index + j] - points[stride + index + j]) * (points[index + j] - points[stride + index + j]);
	}
	//distArray[i] = hypotf((points[index] - points[stride + index]), (points[index + 1] - points[stride + index + 1]));
	distArray[i] = sqrtf(squaredSum);
	__syncthreads(); // sync all the threads before performing a reduction
	
	// reduce
	float sum = 0;
	if (threadIdx.x == 0) {
		for (int k = 0;k < length;k++) {
			sum += distArray[k];
		}
		*distance = sum;
	}
}

__global__ static void bitonicsort(float * values, int lineNos)
{
	extern __shared__ float shared[];
	const unsigned int tid = threadIdx.x;
	shared[tid] = values[tid];
	__syncthreads();
	for (unsigned int k = 2;k <= lineNos ;k *= 2) {
		for (unsigned int j = k / 2; j > 0; j /= 2)
		{
			unsigned int ixj = tid ^ j;

			if (ixj > tid) {
				if ((tid & k) == 0)
				{
					if (shared[tid] > shared[ixj])
					{
						float temp = shared[tid];
						shared[tid] = shared[ixj];
						shared[ixj] = temp;
					}
				}

				else
				{
					if (shared[tid] < shared[ixj])
					{
						float temp = shared[tid];
						shared[tid] = shared[ixj];
						shared[ixj] = temp;
					}
				}
			}
			__syncthreads();
		}
	}
	values[tid] = shared[tid];
}

int main(int argc, char** argv)
{
	// variables used everywhere
	int num_of_stops = 0;
	int num_of_rows = 0;

	// accept commandline arguments or exit if none provided (filename, from the same folder)
	if (argv[1] == NULL || argv[1] == "") {
		printf("Error Reading File\n");
		exit(0);
	}

	// read file using ifstream
	std::ifstream myFile(argv[1]);

	vector<float> values;
	vector<trajectory> trajectories;
	float * d_values;

	/*while (!std::feof(myFile)) {
		std::getline(myFile);
		fscanf(myFile, "%d\n", &values[i]);
		for (int j = 0; j < ) {

		}
	}*/

	string line = "";

	// open file and start parsing line by line
	while (getline(myFile, line))
	{
		trajectory trajTemp;

		stringstream lineTokens(line);
		string temp; // breaking line into string
		vector<int> temp1;
		// parse line and extract numbers from the line
		while (getline(lineTokens, temp, ' ')) {
			temp1.push_back(atoi(temp.c_str()));
		}

		if (2 == temp1.size()) {
			// from the first line, acquire the number of stops
			num_of_rows = temp1[0];
			num_of_stops = temp1[1];
		}
		else {
			// this applies for the lines after the first line
			// calculating stride
			int stride = temp1.size() / num_of_stops;

			// fill in the origin point
			for (int i = 0; i < stride; i++) {
				temp1.insert(temp1.begin(),0);
			}

			// calculating parallel euclidean distance using GPU for the entire trajectory
			// this performs an outer loop parallelization by loop distribution
			// i.e. I broke the Euclidean distance calculation and sorting in 2 different GPU operations
			// this ensures that calculation heavy operations are efficiently performed by the GPU.
			int* d_points;
			float* d_distArray;
			float* d_res;
			hipMalloc(&d_points, temp1.size() * sizeof(int));
			hipMalloc(&d_distArray, num_of_stops * sizeof(float));
			hipMalloc(&d_res, sizeof(float));

			hipMemcpy(d_points, temp1.data(), temp1.size() * sizeof(int), hipMemcpyHostToDevice);
			
			euclidean << <1, num_of_stops >> > (d_points, stride, d_distArray, d_res, num_of_stops);

			hipMemcpy(trajTemp.distance, d_res, sizeof(float), hipMemcpyDeviceToHost);

			hipFree(d_points);
			hipFree(d_distArray);
			hipFree(d_res);

			values.push_back(*trajTemp.distance);
			
			// add all points to the temporary trajectory vector
			trajTemp.trajectoryPoints = temp1;

			// push current trajectory to our trajectories vector
			trajectories.push_back(trajTemp);
		}
	}

	// close the input file
	myFile.close();

	hipMalloc(&d_values, num_of_rows * sizeof(float));
	hipMemcpy(d_values, values.data(), sizeof(float) * num_of_rows, hipMemcpyHostToDevice);

	bitonicsort << <1, num_of_rows, num_of_rows * sizeof(float)>> > (d_values, num_of_rows);

	float* sortedValues = new float[num_of_rows];
	hipMemcpy(sortedValues, d_values, sizeof(float) * num_of_rows, hipMemcpyDeviceToHost);
	
	// uncomment this snippet to debug the code and see output of the sorted array coming from the GPU
	/*printf("\n---------------------------------\n");
	printf("Here is the sorted array from GPU: \n");
	for (int i = 0; i < num_of_rows; i++)
	{
		printf("%f, ", sortedValues[i]);
	}
	printf("\n---------------------------------\n");*/

	// write the output to a file
	std::ofstream output_file("./output.txt");
	for (int i = 0; i < num_of_rows; i++)
	{
		// the trajectories are not sorte in the data structure but we now have the sorted array
		vector<int> trajectoryToPrint;
		for (int j = 0; j < trajectories.size(); j++) {
			if (*trajectories[j].distance == sortedValues[i]) {
				trajectoryToPrint = trajectories[j].trajectoryPoints;
			}
		}
		// write vector to file
		ostream_iterator<int> output_iterator(output_file, " ");
		copy(trajectoryToPrint.begin()+2, trajectoryToPrint.end(), output_iterator);
		output_file << "\n";
	}

	// free memory
	hipFree(d_values);
	free(sortedValues);
}
